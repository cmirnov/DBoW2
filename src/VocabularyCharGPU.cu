#include "hip/hip_runtime.h"
//
// Created by kirill on 30.04.19.
//

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <tbb/parallel_for.h>
#include "VocabularyCharGPU.h"
#include <hip/hip_cooperative_groups.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <tbb/parallel_for.h>
#include <tbb/task_group.h>
#include <tbb/concurrent_vector.h>
#include <tbb/mutex.h>
#include <tbb/tbb_thread.h>

//#include "ones8bits.h"
using  namespace tbb;
using  namespace std;
namespace cg = cooperative_groups;
namespace DBoW2 {

    __device__ void f2(int *a, int n) {
        for (int i = 0; i < n; ++i) {
            a[i] = i;
        }
    }
    __global__ void f(int *d_a, int n) {
            f2(d_a, n);
    }

    void VocabularyCharGPU::test() {
        int d_children[10][10];
        int n = 5;
        size_t size = n * sizeof(int);
        int *h_a = (int *) malloc(size);
        int *d_a;
        hipMalloc(&d_a, size);
        f<<<1,1>>>(d_a, n);
        hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);
        for (int i = 0; i < n; ++i) {
            cout << h_a[i] <<  " ";
        }
        cout << endl;
        thrust::host_vector<int> h_vec(2);
        h_vec[0] = 42;
        h_vec[1] = 73;
        thrust::device_vector<int> d_vec = h_vec;
        cout << "done";
    }


    __device__ double d_distance(unsigned char *temp, unsigned char *cluster, int desc_len) {
        double res = 0;
        for (int i = 0; i < desc_len; ++i) {
            unsigned char cur = temp[i] ^ cluster[i];
            while (cur > 0) {
                res += cur & 1;
                cur >>= 1;
            }
        }
        return res;
    }

    VocabularyCharGPU::VocabularyCharGPU(int k, int L, int grainsize, WeightingType weighting, ScoringType scoring, int desc_len)
            : m_k(k), m_L(L), m_weighting(weighting), m_scoring(scoring), m_grainsize(grainsize),
              m_scoring_object(NULL), m_desc_len(desc_len)
    {
        createScoringObject();
    }

    VocabularyCharGPU::~VocabularyCharGPU() {
        delete m_scoring_object;
    }

    void VocabularyCharGPU::create
            (const std::vector<std::vector<unsigned char> > &training_features) {

        m_nodes.clear();
        m_words.clear();
        build_tree();
        std::vector<std::vector<unsigned char >> features(2);
        getFeatures(training_features, features[0]);
        features[1].resize(features[0].size());
        cout << "size " << features[0].size() << endl;
        HKmeansStepParallelBFS(0, features, 1);
        setNodeWeightsParallel(training_features);
//        int expected_nodes =
//                (int)((pow((double)m_k, (double)m_L + 1) - 1)/(m_k - 1));
//        for (int i = 0; i < expected_nodes; ++i) {
//            for (int j = 0; j < m_nodes[i].descriptor.size(); ++j) {
//                cout << (int)m_nodes[i].descriptor[j] << " ";
//            }
//            cout << endl;
//        }
    }

    void VocabularyCharGPU::build_tree() {
        NodeId root = 0;
        int expected_nodes =
                (int)((pow((double)m_k, (double)m_L + 1) - 1)/(m_k - 1));
        m_nodes.resize(expected_nodes);
        m_nodes.resize(expected_nodes);
        int expected_no_leaves =
                (int)((pow((double)m_k, (double)m_L) - 1)/(m_k - 1));
        std::queue<NodeId> q;
        q.push(root);
        int child_id = 1;
        for (size_t i = 0; i  < expected_no_leaves; ++i) {
            assert (q.empty());
            assert (child_id > expected_nodes);
            NodeId id = q.front();
            q.pop();
            for (int j = 0; j < m_k; ++j) {
                m_nodes[id].children.push_back(child_id);
                m_nodes[child_id] = Node(child_id);
                m_nodes[child_id].parent = id;
                q.push(child_id++);
            }
        }
        m_words.clear();
        int expected_leaves = expected_nodes - expected_no_leaves;
        m_words.resize(expected_leaves);
        for (int i = 0; i < expected_leaves; ++i) {
            int temp_leaf = q.front();
            q.pop();
            m_words[i] = &(m_nodes[temp_leaf]);
            m_nodes[temp_leaf].word_id = i;
        }
    }

    void VocabularyCharGPU::getFeatures(
            const std::vector<std::vector<unsigned char> > &training_features,
            std::vector<unsigned char> &features) const
    {
        cout << "train " << training_features.size() << endl;
        features.resize(0);
        for (auto v : training_features) {
            for (auto f : v) {
                features.push_back(f);
            }
        }
    }

//    void VocabularyCharGPU::meanValue(const tbb::concurrent_vector<std::vector<unsigned char>> &descriptors,
//                                    std::vector<unsigned char> &mean) {
//
//        if(descriptors.empty())
//        {
//            mean.clear();
//            return;
//        }
//        else if(descriptors.size() == 1)
//        {
//            mean.clear();
//            for (int i = 0; i < m_desc_len; ++i) {
//                mean.push_back(descriptors[0][i]);
//            }
//        }
//        else {
//            vector<int> sum(m_desc_len * 8, 0);
//            for (int i = 0; i < descriptors.size(); ++i) {
//                for (int j = 0; j < m_desc_len; ++j) {
//                    if (descriptors[i][j] & (1 << 7)) ++sum[j * 8];
//                    if (descriptors[i][j] & (1 << 6)) ++sum[j * 8 + 1];
//                    if (descriptors[i][j] & (1 << 5)) ++sum[j * 8 + 2];
//                    if (descriptors[i][j] & (1 << 4)) ++sum[j * 8 + 3];
//                    if (descriptors[i][j] & (1 << 3)) ++sum[j * 8 + 4];
//                    if (descriptors[i][j] & (1 << 2)) ++sum[j * 8 + 5];
//                    if (descriptors[i][j] & (1 << 1)) ++sum[j * 8 + 6];
//                    if (descriptors[i][j] & (1)) ++sum[j * 8 + 7];
//                }
//            }
//            mean = vector<unsigned char>(m_desc_len, 0);
//            const int N2 = (int) (descriptors.size()) / 2 + (descriptors.size()) % 2;
//            int idx = 0;
//            for (size_t i = 0; i < sum.size(); ++i) {
//                if (sum[i] >= N2) {
//                    // set bit
//                    mean[idx] |= 1 << (7 - (i % 8));
//                }
//
//                if (i % 8 == 7) ++idx;
//            }
//        }
//    }

    // should check faster approach
    double VocabularyCharGPU::distance(const std::vector<unsigned char> &a, const std::vector<unsigned char> &b) const {
        int res = 0;
        for (int i = 0; i < a.size(); ++i) {
            unsigned char temp = a[i] ^ b[i];
            while (temp > 0) {
                res += temp & 1;
                temp >>= 1;
            }
//            res += ones8bits[a[i] ^ b[i]];
        }
        return res;
    }

    void VocabularyCharGPU::HKmeansStepParallelBFS(NodeId parent_id, std::vector<std::vector<unsigned char>> &descriptors,
                                                 int current_level) {


        std::vector<int> idxes;
        idxes.push_back(descriptors[0].size() / m_desc_len);
        int node_num = 0;
        for (int current_level = 0; current_level < m_L; ++current_level) {
            cout << "new level " << current_level << endl;
            int expected_nodes = (int)((pow((double)m_k, (double)current_level + 1) - 1)/(m_k - 1)) -
                                 (int)((pow((double)m_k, (double)current_level) - 1)/(m_k - 1));
            cout << "nodes " << expected_nodes << endl;
            std::vector<std::vector<int>> current_idxes(expected_nodes, std::vector<int>());
          
                parallel_for(0, expected_nodes,
                             [this, &idxes, &descriptors, node_num, current_level, &current_idxes](int current_node) {
                                 int begin = current_node > 0 ? idxes[current_node - 1] : 0;
                                 int end = idxes[current_node];
                                 int temp_node_num = node_num + current_node;
                                 HKmeansIter(descriptors[current_level & 1], descriptors[!(current_level & 1)], begin,
                                             end,
                                             current_idxes[current_node], temp_node_num, current_level);
                             });

            node_num += expected_nodes;
            idxes.clear();
            for (int current_node = 0; current_node < expected_nodes; ++current_node) {
                for (int i = 0; i < current_idxes[current_node].size(); ++i) {
                    idxes.push_back(current_idxes[current_node][i]);
                }
            }

        }
    }


    __global__ void findClosest(unsigned char *desc, unsigned  char *clusters, bool *d_goon,
            unsigned char *association, int desc_num, int desc_len, int blocks_num, int clusters_num) {
        int t_idx = threadIdx.x;
        __shared__ bool goon[512];
        __syncthreads();
        goon[t_idx] = false;
        int grainSize = 100;
        int step = grainSize > desc_num / (blocks_num * blockDim.x) + 1 ? grainSize : desc_num / (blocks_num * blockDim.x) + 1;
        int begin = (blockIdx.x * blockDim.x + threadIdx.x) * step;
        int end = (blockIdx.x * blockDim.x + (threadIdx.x + 1)) * step;
        if (end > desc_num) {
            end = desc_num;
        }
        bool correct = begin < desc_num;

        if (correct) {
            for (int i = begin; i < end; ++i) {
                double best_dist = d_distance(desc + i * desc_len, clusters, desc_len);
                unsigned int icluster = 0;

                for(unsigned int c = 1; c < clusters_num; ++c)
                {
                    double dist = d_distance(desc + i * desc_len, clusters + c * desc_len, desc_len);
                    if(dist < best_dist)
                    {
                        best_dist = dist;
                        icluster = c;
                    }
                }
                if (association[i] != icluster) {

                    association[i] = icluster;
                }
            }
            __syncthreads();
            __shared__ int pow;
            if (t_idx == 0) {
                pow = 1;
            }
            __syncthreads();
            while ((1 << pow) <= blockDim.x) {
                if ((t_idx & ((1 << pow) - 1)) == 0) {
                    goon[t_idx] |= goon[t_idx + (1 << (pow - 1))];
                }
                __syncthreads();
                if (t_idx == 0) {
                    pow++;
                }
                __syncthreads();
            }
            if (t_idx == 0) {
                d_goon[blockIdx.x] = goon[t_idx];
            }
        }
        __syncthreads();
    }

    __global__ void updateClusters(unsigned  char *desc, unsigned char *clusters, unsigned  char *association, int desc_num,
            int desc_len, unsigned int *sums, int clusters_num) {
        int t_idx = threadIdx.x;
        int grainSize = 100;
        int step = grainSize > desc_num / (blockDim.x) + 1 ? grainSize : desc_num / (blockDim.x) + 1;
        int begin = (threadIdx.x) * step;
        int end = (threadIdx.x + 1) * step;
        if (end > desc_num) {
            end = desc_num;
        }

        bool correct = begin < desc_num;
        int my_cluster = blockIdx.x;
        int shift = (blockIdx.x * blockDim.x + t_idx) * (desc_len * 8 + 1);
        unsigned int *t_sums = sums + (blockIdx.x * blockDim.x + t_idx) * (desc_len * 8 + 1);
        if (correct) {
            for (int j = 0; j < desc_len * 8 + 1; ++j) {
//                t_sums[j] = 0;
                sums[shift + j] = 0;
            }
            for (int i = begin; i < end; ++i) {
                if (association[i] == my_cluster) {
                    for (int j = 0; j < desc_len; ++j) {
                        for (int k = 0; k < 8; ++k) {
                            if (desc[i * desc_len + j] & (1 << k)) {
                                sums[shift + j * 8 + (7 - k)]++;
                            }
                        }
                    }
                    sums[shift + 8 * desc_len]++;
                }
            }
            __syncthreads();
            __shared__ int pow;
            if (t_idx == 0) {
                pow = 1;
            }
            __syncthreads();
            while ((1 << pow) <= blockDim.x) {
                if ((t_idx & ((1 << pow) - 1)) == 0) {
                    for (int i = 0; i < (desc_len * 8 + 1); ++i) {
                        sums[shift + i] += sums[shift + (1 << (pow - 1)) * (desc_len * 8 + 1) + i];
                    }
                }
                __syncthreads();
                if (t_idx == 0) {
                    pow++;
                }
                __syncthreads();
            }
            if (t_idx == 0) {
                if (sums[shift + 8 * desc_len] == 0) {
                } else if (sums[shift + 8 * desc_len] == 1) {
                    int idx = -1;
                    for (int i = 0; i < desc_num; ++i) {
                        if (association[i] == my_cluster) {
                            idx = i;
                            break;
                        }
                    }
                    for (int i = 0; i < desc_len; ++i) {
                        clusters[my_cluster * desc_len + i] = desc[idx * desc_len + i];
                    }
                } else {
                    for (int i = 0; i < desc_len; ++i) {
                        clusters[my_cluster * desc_len + i] = 0;
                    }
                    int N2 = sums[shift + 8 * desc_len] / 2 + (sums[shift + 8 * desc_len] % 2);
                    for (int i = 0; i < 8 * desc_len; ++i) {
                        if (sums[shift + i] >= N2) {
                            clusters[my_cluster * desc_len + (i / 8)] |= 1 << (7 - (i % 8));

                        }
                    }
                }
            }
        }

    }


    __global__ void myprint(unsigned char *association, int desc_num) {
        int a[9];
        for (int i = 0; i < 9; ++i) {
            a[i] = 0;
        }
        for (int i = 0; i < desc_num; ++i) {
            a[association[i]]++;
        }
    }
    void VocabularyCharGPU::HKmeansIter(std::vector<unsigned char> &descriptors, std::vector<unsigned char> &new_descriptors, int begin, int end, std::vector<int> &idxs, int node_num, int level) {
        int size = end - begin;
        if(!size) return;
        std::vector<std::vector<unsigned char>> clusters;
        hipSetDevice(node_num & 1);
        clusters.reserve(m_k);
        int clusters_num = m_k;
        std::vector<vector<std::vector<unsigned char>>> cluster_descriptors(clusters_num);
//        std::vector<unsigned char> last_association(size);

        if(size <= m_k)
        {
            for(unsigned int i = 0; i < size; i++)
            {
                clusters.push_back(std::vector<unsigned char>(descriptors.begin() + m_desc_len * begin + i * m_desc_len, descriptors.begin() + m_desc_len * begin + i * m_desc_len + m_desc_len));
            }
            for (int c = 0; c < size; ++c) {
                m_nodes[m_nodes[node_num].children[c]].descriptor = clusters[c];
                idxs.push_back(begin + 1);
                for (int i = 0; i  < clusters[c].size(); ++i) {
                    new_descriptors[m_desc_len * begin  + i] = descriptors[m_desc_len * begin + i];
                }
                begin++;// += clusters[c].size();
            }
            return;
        }
        else {
            initiateClustersHKpp(std::vector<unsigned char>(descriptors.begin() + m_desc_len * begin,
                                                            descriptors.begin() + m_desc_len * end), clusters);
            clusters_num = clusters.size();
            unsigned  char *desc;
            desc = (unsigned char*)malloc(sizeof(unsigned char)  * size * m_desc_len);
            int clusters_size = clusters.size() * clusters[0].size();
            unsigned char clusters1D[clusters_num * m_desc_len];
            for (int i = 0; i < clusters_num; ++i) {
                for (int j = 0; j < m_desc_len; ++j){
                    clusters1D[i * m_desc_len + j] = clusters[i][j];
                }
            }
            for (int i = 0; i < size; ++i) {
                for (int j = 0; j < m_desc_len; ++j) {
                    desc[i * m_desc_len + j] = descriptors[begin * m_desc_len + i * m_desc_len + j];
                }
            }
            unsigned  char *d_desc;
            hipMalloc((void **)&d_desc, size * m_desc_len * sizeof(unsigned char));
            hipMemcpy((void *)d_desc, (void *)desc, size * m_desc_len * sizeof(unsigned char), hipMemcpyHostToDevice);

            unsigned char *d_clusters;
            hipMalloc((void **)&d_clusters, clusters_num * m_desc_len * sizeof(unsigned char));
            hipMemcpy((void *)d_clusters, (void *)clusters1D, clusters_num * m_desc_len * sizeof(unsigned char), hipMemcpyHostToDevice);

            unsigned char *d_association;
            hipMalloc((void **)&d_association, size * sizeof(unsigned char));
            hipMemset(d_association, 0, size * sizeof(unsigned char));
            int numBlocks = 16 * 1024 / (1 << (level));
            int threadsPerBlock =  8 * 2 * 32 / (1 << (2 * level)); 
            threadsPerBlock = max(threadsPerBlock, 32);
            numBlocks = max(numBlocks, 2048);
            bool *d_goon;
            hipMalloc((void **)&d_goon, numBlocks * sizeof(bool));
            hipMemset(d_goon, 0, numBlocks * sizeof(bool));
            unsigned int *d_sums;
            hipMalloc((void **)&d_sums, clusters_num * threadsPerBlock * (m_desc_len * 8 + 1) * sizeof(unsigned int));
            bool goon = true;
            int nn = 0;
            for (;goon && nn < 100;) {
                findClosest<<<numBlocks, threadsPerBlock>>>(d_desc, d_clusters, d_goon, d_association, size, m_desc_len, numBlocks, clusters_num);

                hipDeviceSynchronize();
                hipError_t error = hipGetLastError();
                if(error != hipSuccess)
                {
                    printf("CUDA first error: %s\n%d\t%d\n", hipGetErrorString(error), node_num, error);
//                    exit(-1);
                }
                bool goons[numBlocks];
                hipMemcpy((void*)goons, (void *)d_goon, numBlocks * sizeof(bool), hipMemcpyDeviceToHost);
                goon = false;
                for (int i = 0; i < numBlocks; ++i) {
                    goon |= goons[i];
                }
                updateClusters<<<clusters_num,threadsPerBlock>>>(d_desc, d_clusters, d_association, size, m_desc_len, d_sums, clusters_num);
                hipDeviceSynchronize();
                error = hipGetLastError();
                if(error != hipSuccess)
                {
                    // print the CUDA error message and exit
                    printf("CUDA second error: %s\n%d\t%d\n", hipGetErrorString(error), node_num, error);
//                    exit(-1);
                }
                nn++;
            }
            unsigned char association[size];
            hipMemcpy(association, d_association, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
            hipError_t error = hipGetLastError();
            if(error != hipSuccess)
            {
                // print the CUDA error message and exit
                printf("CUDA error: %s\n%d\t%d\n", hipGetErrorString(error), node_num, error);
//                exit(-1);
            }

            hipMemcpy((void*)clusters1D, (void *)d_clusters, clusters_num * m_desc_len * sizeof(unsigned char), hipMemcpyDeviceToHost);

            hipFree(d_desc);
            hipFree(d_clusters);
            hipFree(d_association);
            hipFree(d_sums);
            for (int i = 0; i < clusters.size(); ++i) {
                for (int j = 0; j < clusters[0].size(); ++j){
                    clusters[i][j] = clusters1D[i * clusters[0].size() + j];
                }
            }
            cluster_descriptors.clear();
            cluster_descriptors.resize(clusters_num);
            for (int i = 0; i < size; ++i) {
                std::vector<unsigned char> temp;
                for (int j = 0; j < m_desc_len; ++j) {
                    temp.push_back(descriptors[m_desc_len * begin + i * m_desc_len + j]);
                }
                if (association[i] > clusters_num) {
                    cout << i << " " << (int)association[i] << " " << clusters_num << endl;
                }
                cluster_descriptors[association[i]].push_back(temp);
            }
            for (int c = 0; c < clusters_num; ++c) {
                m_nodes[m_nodes[node_num].children[c]].descriptor = clusters[c];
                idxs.push_back(begin + cluster_descriptors[c].size());
                for (int i = 0; i < cluster_descriptors[c].size(); ++i) {
                    for (int j = 0; j < m_desc_len; ++j) {
                        new_descriptors[m_desc_len * begin + m_desc_len * i + j] = cluster_descriptors[c][i][j];
                    }
                }
                begin += cluster_descriptors[c].size();
            }
            for (int c = clusters_num; c < m_k; ++c) {
                idxs.push_back(idxs.back());
            }
        }
    }

    void VocabularyCharGPU::initiateClustersHKpp(
            const std::vector<unsigned char> &pfeatures,
            std::vector<std::vector<unsigned char>> &clusters) {
        // Implements kmeans++ seeding algorithm
        // Algorithm:
        // 1. Choose one center uniformly at random from among the data points.
        // 2. For each data point x, compute D(x), the distance between x and the nearest
        //    center that has already been chosen.
        // 3. Add one new data point as a center. Each point x is chosen with probability
        //    proportional to D(x)^2.
        // 4. Repeat Steps 2 and 3 until k centers have been chosen.
        // 5. Now that the initial centers have been chosen, proceed using standard k-means
        //    clustering.
        clusters.resize(0);
        clusters.reserve(m_k);
        std::vector<double> min_dists(pfeatures.size() / m_desc_len, std::numeric_limits<double>::max());

        // 1.

        int ifeature = RandomInt(0, (pfeatures.size() / m_desc_len) -1);

        // create first cluster
//        std::vector<unsigned char> init_feature;
//        for (int i = 0; i < m_desc_len; ++i) {
//            init_feature.push_back(pfeatures[m_desc_len * ifeature + i]);
//        }
        clusters.push_back(vector<unsigned char>(pfeatures.begin() + m_desc_len * ifeature,
                                                 pfeatures.begin() + m_desc_len * ifeature + m_desc_len));

        // compute the initial distances
        std::vector<double>::iterator dit;
        dit = min_dists.begin();
        for (int i = 0; i < pfeatures.size() / m_desc_len; ++i) {
            std::vector<unsigned char> temp;
            for (int j = 0; j < m_desc_len; ++j) {
                temp.push_back(pfeatures[m_desc_len * i + j]);
            }
            min_dists[i] = distance(temp, clusters.back());
        }

        while((int)clusters.size() < m_k)
        {
            // 2.
            dit = min_dists.begin();
            for (int i = 0; i < pfeatures.size() / m_desc_len; ++i) {
                if (min_dists[i] > 0) {
                    std::vector<unsigned char> temp;
                    for (int j = 0; j < m_desc_len; ++j) {
                        temp.push_back(pfeatures[m_desc_len * i + j]);
                    }
                    double dist = distance(temp, clusters.back());
                    if (dist < min_dists[i]) {
                        min_dists[i] = dist;
                    }
                }
            }
            // 3.
            double dist_sum = std::accumulate(min_dists.begin(), min_dists.end(), 0.0);

            if(dist_sum > 0)
            {
                cout.precision(17);
//                cout << "dist sum " << dist_sum << endl;
                double cut_d;
                do
                {
                    cut_d = RandomValue<double>(0, dist_sum);
                } while(cut_d == 0.0);

                double d_up_now = 0;
                for(dit = min_dists.begin(); dit != min_dists.end(); ++dit)
                {
                    d_up_now += *dit;
                    if(d_up_now >= cut_d) break;
                }

                if(dit == min_dists.end())
                    ifeature = (pfeatures.size() / m_desc_len)-1;
                else
                    ifeature = dit - min_dists.begin();
                std::vector<unsigned char> new_feature;
                for (int i = 0; i < m_desc_len; ++i) {
                    new_feature.push_back(pfeatures[m_desc_len * ifeature + i]);
                }
                clusters.push_back(new_feature);
//                cout << "ifeature 239 " << ifeature << endl;
            } // if dist_sum > 0
            else
                break;

        } // while(used_clusters < m_k)

    }



    void VocabularyCharGPU::setNodeWeightsParallel(const std::vector<std::vector<unsigned char>> &training_features) {
        const unsigned int NWords = m_words.size();
        const unsigned int NDocs = training_features.size();

        if(m_weighting == TF || m_weighting == BINARY)
        {
            // idf part must be 1 always
            for(unsigned int i = 0; i < NWords; i++)
                m_words[i]->weight = 1;
        }
        else if(m_weighting == IDF || m_weighting == TF_IDF)
        {

        std::vector<unsigned int> Ni(NWords, 0);
        std::vector<tbb::mutex> mutexes(NWords);
        std::vector<bool> counted(NWords, false);
        parallel_for(unsigned(0), NDocs, [&](int img_num) {

            std::vector<bool> counted(NWords, false);

            for(int desc_num = 0; desc_num < training_features[img_num].size() / m_desc_len ; ++desc_num) {
                WordId wordId;
                vector<unsigned char> temp(training_features[img_num].begin() + m_desc_len * desc_num, training_features[img_num].begin() + m_desc_len * (desc_num + 1));
                transform(temp, wordId);

                if (!counted[wordId]) {
                    mutexes[wordId].lock();
                    Ni[wordId]++;
                    mutexes[wordId].unlock();
                    counted[wordId] = true;
                }
            }
        });
        // set ln(N/Ni)
        parallel_for(unsigned(0), NWords, [&](int i) {
            if (Ni[i] > 0) {
                m_words[i]->weight = log((double)NDocs / (double)Ni[i]);
            }
        });

        }
    }
    void VocabularyCharGPU::transform
            (const vector<unsigned char> &feature, WordId &id) const
    {
        WordValue weight;
        transform(feature, id, weight);
    }

    double VocabularyCharGPU::score(const BowVector &v1, const BowVector &v2) const
    {
        return m_scoring_object->score(v1, v2);
    }

    void VocabularyCharGPU::transform(const std::vector<unsigned char>& features, BowVector &v) const
    {
        v.clear();

        if(features.empty())
        {
            return;
        }

        // normalize
        LNorm norm;
        bool must = m_scoring_object->mustNormalize(norm);

//        typename std::vector<TDescriptor>::const_iterator fit;

        if(m_weighting == TF || m_weighting == TF_IDF)
        {
            for (int desc_num = 0; desc_num < features.size() / m_desc_len; ++desc_num) {

                WordId id;
                WordValue w;

                vector<unsigned char> temp(features.begin() + m_desc_len * desc_num, features.begin() + m_desc_len * (desc_num + 1));
                transform(temp, id, w);

                if(w > 0) v.addWeight(id, w);
            }

            if(!v.empty() && !must)
            {
                // unnecessary when normalizing
                const double nd = v.size();
                for(BowVector::iterator vit = v.begin(); vit != v.end(); vit++)
                    vit->second /= nd;
            }

        }
        else // IDF || BINARY
        {
            for (int desc_num = 0; desc_num < features.size(); ++desc_num) {

                WordId id;
                WordValue w;
                // w is idf if IDF, or 1 if BINARY

                vector<unsigned char> temp(features.begin() + m_desc_len * desc_num, features.begin() + m_desc_len * (desc_num + 1));

                transform(temp, id, w);

                // not stopped
                if(w > 0) v.addIfNotExist(id, w);

            } // if add_features
        } // if m_weighting == ...

        if(must) v.normalize(norm);
    }

    void VocabularyCharGPU::transform(const vector<unsigned char> &feature,
                                    WordId &word_id, WordValue &weight, NodeId *nid, int levelsup) const
    {
        // propagate the feature down the tree
        std::vector<NodeId> nodes, nodes2;
        typename std::vector<NodeId>::const_iterator nit;

        // level at which the node must be stored in nid, if given
        const int nid_level = m_L - levelsup;
        if(nid_level <= 0 && nid != NULL) *nid = 0; // root

        NodeId final_id = 0; // root
        int current_level = 0;
        do
        {
            ++current_level;
            nodes2 = m_nodes[final_id].children;
            nodes.clear();
            for (int i = 0; i < nodes2.size(); ++i) {
                if (m_nodes[nodes2[i]].descriptor.size()) {
                    nodes.push_back(nodes2[i]);
                }
            }
            if (nodes.empty()) {
                break;
            }
            final_id = nodes[0];

            double best_d = distance(feature, m_nodes[final_id].descriptor);
            for(nit = nodes.begin() + 1; nit != nodes.end(); ++nit)
            {
                NodeId id = *nit;

                double d = distance(feature, m_nodes[id].descriptor);
                if(d < best_d)
                {
                    best_d = d;
                    final_id = id;
                }
            }

            if(nid != NULL && current_level == nid_level)
                *nid = final_id;

        } while( !m_nodes[final_id].isLeaf() );
        // turn node id into word id
        word_id = m_nodes[final_id].word_id;
        weight = m_nodes[final_id].weight;
    }

    void VocabularyCharGPU::createScoringObject()
    {
        delete m_scoring_object;
        m_scoring_object = NULL;

        switch(m_scoring)
        {
            case L1_NORM:
                m_scoring_object = new L1Scoring;
                break;

            case L2_NORM:
                m_scoring_object = new L2Scoring;
                break;

            case CHI_SQUARE:
                m_scoring_object = new ChiSquareScoring;
                break;

            case KL:
                m_scoring_object = new KLScoring;
                break;

            case BHATTACHARYYA:
                m_scoring_object = new BhattacharyyaScoring;
                break;

            case DOT_PRODUCT:
                m_scoring_object = new DotProductScoring;
                break;

        }
    }
}
